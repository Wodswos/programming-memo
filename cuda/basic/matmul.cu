#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

#define BLOCK_SIZE 16

// void printMatrix(const float* matrix, int N, const char* name){
//     const int print_size = 2048;
//     for (int i = 0; i < print_size && i < N; i++){
//         for (int j = 0; j < print_size && j < N; j++){
//             printf("%6.2f ", matrix[i * N + j]);
//         }
//         printf("\n");
//     }
//     printf("\n");
// }

__global__ void matrixMul(float *C, const float* A, const float* B, int N){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    C[row * N + col] = 0.0f;
    // 就爱写点 python.
    if (row < N && col < N)
        for (size_t i = 0; i < N; i++) 
            C[row * N + col] += A[row * N + i] * B[i * N + col];
}

int main(int argc, char **argv){

    // TODO:
    // 1. 用户自定义矩阵大小
    // 2. bind 到 python api

    int N = 4096 * 8;
    size_t size = N * N * sizeof(float);

    // 分配 host 内存
    float* h_A = (float*) malloc(size);
    float* h_B = (float*) malloc(size);
    float* h_C = (float*) malloc(size);

    // init a random matrix
    for (int i = 0; i < N * N; ++i){
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // 分配 deivce 内存
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // host -> device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // caclulate
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((N + dimBlock.x - 1)/dimBlock.x, (N + dimBlock.y -1)/dimBlock.y);
    auto start_time = std::chrono::high_resolution_clock::now();

    matrixMul<<<dimGrid, dimBlock>>>(d_C, d_A, d_B, N);

    hipDeviceSynchronize();
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end_time - start_time;
    printf("Time taken: %f seconds\n", elapsed.count());
    // 0.178791 seconds on 3090 for N = 4096
    // 1.359545 seconds on 3090 for N = 8192
    // 10.912323 seconds on 3090 for N = 16384
    // 88.934003 seconds on 3090 for N = 32768

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // print & verify
    // first row of left matrix
    // for (size_t i = 0; i < N; i++) printf("%6.2f ", h_A[i]);
    // printf("\n");
    // first column of right matrix
    // for (size_t i = 0; i < N; i++) printf("%6.2f ", h_B[i * N]);
    // printf("\n");
    // first element of result
    std::cout << "result[0][0]: " << h_C[0] << std::endl;

    // free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}